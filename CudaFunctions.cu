#include "hip/hip_runtime.h"

template <typename T = int>
__global__ void CudaMultKernel(struct mat<T> A, struct mat<T> B, struct mat<T> C)
{
  T Cvalue = 0;
  int row = blockIdx.y * blockDim.y + threadIdx.x;
  int column = blockIdx.x * blockDim.x + threadIdx.y;

  for(int i=0; i<A.width; i++){
    Cvalue += A.elements[row * A.width + i] * B.elements[i * B.width + column];
  }

  C.elements[row * C.width + column] = Cvalue;
}
//__global__ void TiledCudaMultKernel();

template <typename T = int>
double CudaMult(Matrix<T>& A, Matrix<T>& B, Matrix<T>& C, const int bSize) 
//bSize deve essere potenza di due, o meglio è preferibile che lo sia
{
  using namespace std;

  Matrix<T> CA = A.ForceAddTilingPaddingRows(bSize);
  Matrix<T> CB = B.ForceAddTilingPaddingColumns(bSize);
  Matrix<T> CC = C.ForceAddTilingPadding(bSize);
  struct mat<T> h_A, h_B, h_C;

  //h for host
  h_A.width = CA.Columns();
  h_A.height = CA.Rows();
  h_A.padd_height = CA.PaddingRows();
  h_A.padd_width = CA.PaddingColumns();
  h_B.width = CB.Columns();
  h_B.height = CB.Rows();
  h_B.padd_height = CB.PaddingRows();
  h_B.padd_width = CB.PaddingColumns();
  h_C.width = CC.Columns();
  h_C.height = CC.Rows();
  h_C.padd_height = CC.PaddingRows();
  h_C.padd_width = CC.PaddingColumns();

  h_A.elements = new T[CA.Rows()*CA.Columns()];
  h_B.elements = new T[CB.Rows()*CB.Columns()];
  h_C.elements = new T[CC.Rows()*CC.Columns()];

  CA.BlurtMatrix(h_A.elements);
  CB.BlurtMatrix(h_B.elements);
  CC.BlurtMatrix(h_C.elements);

  for(int i=0; i<CA.Rows()*CA.Columns(); i++){
    cout<<CA.GetElement(i/CA.Columns(), i%CA.Columns())
        << "-vs-"
      <<h_A.elements[i]
      <<"\t";
  }

  clock_t tic = clock();


  size_t Asize = h_A.height * h_A.width * sizeof(TYPE);
  size_t Bsize = h_B.height * h_B.width * sizeof(TYPE);
  size_t Csize = h_C.height * h_C.width * sizeof(TYPE);

  //now to allocate GPU memory
  struct mat<T> d_A;
  d_A.width = h_A.width;
  d_A.height = h_A.height;
  d_A.padd_height = h_A.padd_height;
  d_A.padd_width = h_A.padd_width;
  hipMalloc(&d_A.elements, Asize);

  struct mat<T> d_B;
  d_B.width = h_B.width;
  d_B.height = h_A.height;
  d_B.padd_height = h_B.padd_height;
  d_B.padd_width = h_B.padd_width;
  hipMalloc(&d_B.elements, Bsize);

  struct mat<T> d_C;
  d_C.width = h_C.width;
  d_C.height = h_A.height;
  d_C.padd_height = h_C.padd_height;
  d_C.padd_width = h_C.padd_width;
  hipMalloc(&d_C.elements, Csize);

  //now to populate the memory
  hipMemcpy(d_A.elements, h_A.elements, Asize, hipMemcpyHostToDevice);
  hipMemcpy(d_B.elements, h_B.elements, Bsize, hipMemcpyHostToDevice);
  //C will be populated when it's calculated

  dim3 dimBlock(bSize, bSize);
  dim3 dimGrid(B.Columns() / dimBlock.x, A.Rows() / dimBlock.y);
  CudaMultKernel<<< dimGrid, dimBlock >>>(d_A, d_B, d_C);

  hipMemcpy(h_C.elements, d_C.elements, Csize, hipMemcpyDeviceToHost);

  clock_t toc = clock();

  CC.InitMatrix(h_C.elements, h_C.height * h_C.width);

  double execution_time = (double)(toc-tic)/CLOCKS_PER_SEC;

  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);

  delete h_A.elements;
  delete h_B.elements;
  delete h_C.elements;

  return execution_time;
}//double TiledCudaMult();
